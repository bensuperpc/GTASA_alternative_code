#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////
//   ____                                                   //
//  | __ )  ___ _ __  ___ _   _ _ __   ___ _ __ _ __   ___  //
//  |  _ \ / _ \ '_ \/ __| | | | '_ \ / _ \ '__| '_ \ / __| //
//  | |_) |  __/ | | \__ \ |_| | |_) |  __/ |  | |_) | (__  //
//  |____/ \___|_| |_|___/\__,_| .__/ \___|_|  | .__/ \___| //
//                             |_|             |_|          //
//////////////////////////////////////////////////////////////
//                                                          //
//  BenLib, 2021                                            //
//  Created: 21, March, 2021                                //
//  Modified: 29, April, 2022                               //
//  file: kernel.cu                                         //
//  Crypto                                                  //
//  Source:
//  https://stackoverflow.com/questions/13553015/cuda-c-linker-error-undefined-reference
//  //
//          https://www.olcf.ornl.gov/tutorials/cuda-vector-addition/ //
//          https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#asynchronous-transfers-and-overlapping-transfers-with-computation__concurrent-copy-and-execute
//          https://www.ce.jhu.edu/dalrymple/classes/602/Class12.pdf //
//          https://create.stephan-brumme.com/crc32/
//  CPU: ALL                                                //
//                                                          //
//////////////////////////////////////////////////////////////

#include "wrapper.hpp"

__host__ void jamcrc_wrapper(dim3* grid,
                             dim3* threads,
                             hipStream_t* stream,
                             const int device,
                             const void* data,
                             const uint64_t length,
                             uint32_t* result,
                             const uint32_t previousCrc32)
{
  jamcrc_kernel_wrapper<<<*grid, *threads, device, *stream>>>(data, result, length, previousCrc32);
}

__host__ uint32_t my::cuda::jamcrc(const void* data,
                                   const uint64_t length,
                                   const uint32_t previousCrc32,
                                   const uint cuda_block_size)
{
  int device = 0;
  hipGetDevice(&device);

  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  // Calculate length of the array with max_range and min_range
  uint64_t data_size = (length + 1) * sizeof(void*);
  uint32_t* data_cuda = nullptr;

  uint64_t result_size = 1 * sizeof(uint32_t);
  uint32_t* result_cuda = nullptr;

  hipMallocManaged(&data_cuda, data_size, hipMemAttachGlobal);
  hipMallocManaged(&result_cuda, result_size, hipMemAttachGlobal);

  hipStreamAttachMemAsync(stream, &data_cuda);
  hipStreamAttachMemAsync(stream, &result_cuda);

  hipMemPrefetchAsync(data_cuda, data_size, device, stream);
  hipMemPrefetchAsync(result_cuda, result_size, device, stream);

  // std::copy(data, data + length, data_cuda);
  memcpy(data_cuda, data, data_size);
  *result_cuda = 0;

  uint64_t grid_size = static_cast<uint64_t>(ceil(static_cast<double>(data_size) / cuda_block_size));
  // std::cout << "grid_size: " << static_cast<double>(data_size) / cuda_block_size << std::endl;

  dim3 threads(static_cast<uint>(cuda_block_size), 1, 1);
  dim3 grid(static_cast<uint>(grid_size), 1, 1);

  jamcrc_kernel_wrapper<<<grid, threads, device, stream>>>(data_cuda, result_cuda, length, previousCrc32);

  hipStreamSynchronize(stream);
  hipDeviceSynchronize();

  // std::cout << "result_cuda: " << *result_cuda << std::endl;

  hipFree(data_cuda);
  hipStreamDestroy(stream);

  return *result_cuda;
}

__host__ void my::cuda::launch_kernel(std::vector<uint32_t>& jamcrc_results,
                                      std::vector<uint64_t>& index_results,
                                      const uint64_t min_range,
                                      const uint64_t max_range,
                                      const uint64_t cuda_block_size)
{
  // int device = -1;
  // hipGetDevice(&device);

  int device = 0;
  hipGetDevice(&device);

  /*
  int priority_high, priority_low;
  hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);
  hipStream_t st_high, st_low;
  hipStreamCreateWithPriority(&st_high, hipStreamNonBlocking, priority_high);
  hipStreamCreateWithPriority(&st_low, hipStreamNonBlocking, priority_low);
  */

  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  // hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024);

  // Calculate length of the array with max_range and min_range
  uint64_t array_length = static_cast<uint64_t>((max_range - min_range) / 20000000 + 1);
  uint64_t jamcrc_results_size = array_length * sizeof(uint32_t);
  uint64_t index_results_size = array_length * sizeof(uint64_t);

  uint32_t* jamcrc_results_ptr = nullptr;
  uint64_t* index_results_ptr = nullptr;

  hipMallocManaged(&jamcrc_results_ptr, jamcrc_results_size, hipMemAttachGlobal);
  hipMallocManaged(&index_results_ptr, index_results_size, hipMemAttachGlobal);

  hipStreamAttachMemAsync(stream, &jamcrc_results_ptr);
  hipStreamAttachMemAsync(stream, &index_results_size);

  hipMemPrefetchAsync(jamcrc_results_ptr, jamcrc_results_size, device, stream);
  hipMemPrefetchAsync(index_results_ptr, index_results_size, device, stream);

  for (uint64_t i = 0; i < array_length; ++i) {
    jamcrc_results_ptr[i] = 0;
    index_results_ptr[i] = 0;
  }

  uint64_t grid_size = static_cast<uint64_t>(ceil(static_cast<double>(max_range - min_range) / cuda_block_size));
  std::cout << "CUDA Grid size: " << grid_size << std::endl;
  std::cout << "CUDA Block size: " << cuda_block_size << std::endl;

  dim3 threads(static_cast<uint>(cuda_block_size), 1, 1);
  dim3 grid(static_cast<uint>(grid_size), 1, 1);

  runner_kernel<<<grid, threads, device, stream>>>(
      jamcrc_results_ptr, index_results_ptr, array_length, min_range, max_range);

  jamcrc_results.reserve(array_length);
  index_results.reserve(array_length);

  hipStreamSynchronize(stream);

  for (uint64_t i = 0; i < array_length; ++i) {
    if (jamcrc_results_ptr[i] != index_results_ptr[i]) {
      jamcrc_results.emplace_back(jamcrc_results_ptr[i]);
      index_results.emplace_back(index_results_ptr[i]);
    }
  }

  hipDeviceSynchronize();
  hipFree(jamcrc_results_ptr);
  hipFree(index_results_ptr);

  hipStreamDestroy(stream);
  // hipStreamDestroy(st_high);
  // hipStreamDestroy(st_low);
}

__host__ void my::cuda::launch_kernel(size_t grid,
                                      size_t threads,
                                      hipStream_t& stream,
                                      const int device,
                                      uint32_t* crc_result,
                                      uint64_t* index_result,
                                      uint64_t array_size,
                                      uint64_t a,
                                      uint64_t b)
{
  runner_kernel<<<grid, threads, device, stream>>>(crc_result, index_result, array_size, a, b);
}

__host__ void my::cuda::launch_kernel(dim3& grid,
                                      dim3& threads,
                                      hipStream_t& stream,
                                      const int device,
                                      uint32_t* crc_result,
                                      uint64_t* index_result,
                                      uint64_t array_size,
                                      uint64_t a,
                                      uint64_t b)
{
  runner_kernel<<<grid, threads, device, stream>>>(crc_result, index_result, array_size, a, b);
}
