#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////
//   ____                                                   //
//  | __ )  ___ _ __  ___ _   _ _ __   ___ _ __ _ __   ___  //
//  |  _ \ / _ \ '_ \/ __| | | | '_ \ / _ \ '__| '_ \ / __| //
//  | |_) |  __/ | | \__ \ |_| | |_) |  __/ |  | |_) | (__  //
//  |____/ \___|_| |_|___/\__,_| .__/ \___|_|  | .__/ \___| //
//                             |_|             |_|          //
//////////////////////////////////////////////////////////////
//                                                          //
//  BenLib, 2021                                            //
//  Created: 21, March, 2021                                //
//  Modified: 29, April, 2022                               //
//  file: kernel.cu                                         //
//  Crypto                                                  //
//  Source:
//  https://stackoverflow.com/questions/13553015/cuda-c-linker-error-undefined-reference
//  //
//          https://www.olcf.ornl.gov/tutorials/cuda-vector-addition/ //
//          https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#asynchronous-transfers-and-overlapping-transfers-with-computation__concurrent-copy-and-execute
//          https://www.ce.jhu.edu/dalrymple/classes/602/Class12.pdf //
//          https://create.stephan-brumme.com/crc32/
//  CPU: ALL                                                //
//                                                          //
//////////////////////////////////////////////////////////////

#include "kernel.cuh"

__global__ void jamcrc_kernel_wrapper(const void* data,
                                      uint32_t* result,
                                      const uint64_t length,
                                      const uint32_t previousCrc32)
{
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx == 0) {
    *result = jamcrc_kernel(data, length, previousCrc32);
  }
}

__device__ uint32_t jamcrc_kernel(const void* data, uint64_t length, const uint32_t previousCrc32)
{
  uint32_t crc = ~previousCrc32;
  unsigned char* current = (unsigned char*)data;
  while (length--)
    crc = (crc >> 8) ^ crc32_lookup[(crc & 0xFF) ^ *current++];
  return crc;
}

__global__ void runner_kernel(uint32_t* crc_result, uint64_t* index_result, uint64_t array_size, uint64_t a, uint64_t b)
{
  const uint64_t id = blockIdx.x * blockDim.x + threadIdx.x + a;
  // printf("blockIdx %d, blockDimx %d, threadIdx %d\n", blockIdx.x, blockDim.x,
  // threadIdx.x);

  if (id <= b && id >= a) {
    // printf("blockIdx %d, blockDim %d, threadIdx %d\n", blockIdx.x,
    // blockDim.x, threadIdx.x);

    // Allocate memory for the array
    unsigned char array[29] = {0};

    uint64_t size = 0;
    // Generate the array
    find_string_inv_kernel(array, id, size);

    // Calculate the CRC
    const uint32_t result = jamcrc_kernel(array, size, 0);

    bool found = false;
    for (uint32_t i = 0; i < 87; i++) {
      if (result == cheat_list[i]) {
        found = true;
        break;
      }
    }

    if (!found) {
      return;
    }

    // Todo: Avoid datarace
    //__syncthreads();

    for (uint64_t i = 0; i < array_size; i++) {
      if (crc_result[i] == 0 && index_result[i] == 0) {
        crc_result[i] = result;
        index_result[i] = id;
        break;
      }
    }
  }
}

__device__ void find_string_inv_kernel(unsigned char* array, uint64_t n, uint64_t& terminator_index)
{
  const uint32_t string_size_alphabet = 27;

  const unsigned char alpha[string_size_alphabet] = {"ABCDEFGHIJKLMNOPQRSTUVWXYZ"};
  // If n < 27
  if (n < 26) {
    array[0] = alpha[n];
    array[1] = '\0';
    terminator_index = 1;
    return;
  }
  // If n > 27
  uint64_t i = 0;
  while (n > 0) {
    array[i] = alpha[(--n) % 26];
    n /= 26;
    ++i;
  }
  array[i] = '\0';
  terminator_index = i;
}
